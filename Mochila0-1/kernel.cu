
#ifndef __CUDACC__
#define __CUDACC__
#endif
#include "hip/hip_runtime.h"


#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include <string.h>

#define WEI 11
#define ITEN 5

void inserirPeso(int vet[]);
void inserirValor(int vet[]);
void info(int tam, int pes[], int val[], int n);

hipError_t mochilaWithCuda(int *mochila, const int *peso, const int *valor);

__global__ void mochilaKernel(int *mochila, const int *peso, const int *valor,const int wei,const int iten)
{
	int i;
	int w = threadIdx.x + 1;
	if (w<wei+1){
		for (i = 1; i<iten + 1; i++){
			if (peso[i]>w){
				mochila[i*(wei + 1) + w] = mochila[(i - 1)*(wei + 1) + w];

			}
			else{
				if (mochila[(i - 1)*(wei + 1) + w] > valor[i] + mochila[(i - 1)*(wei + 1) + w - peso[i]]){
					mochila[i*(wei + 1) + w] = mochila[(i - 1)*(wei + 1) + w];

				}
				else{
					mochila[i*(wei + 1) + w] = valor[i] + mochila[(i - 1)*(wei + 1) + w - peso[i]];
				}

			}
		}
	}
}

int main()
{
    
	//declara��o do peso limite da mochila, o numero de itens e variavel auxiliar
	int i, j, w;



	//decla��o do peso e valor de cada item, e da matriz mochila
	int peso[ITEN], valor[ITEN];

	int *mochila;
	mochila = (int*)malloc((ITEN + 1)*(WEI + 1)*sizeof(int));
	for (i = 0; i < (ITEN + 1)*(WEI + 1); i++) mochila[i] = 0;


	//inser��es dos itens com seus respectivos valores
	inserirPeso(peso);
	inserirValor(valor);

	info(WEI, peso, valor, ITEN);

	//Pseudo Codigo Transcrito


    // Add vectors in parallel.
    hipError_t cudaStatus = mochilaWithCuda(mochila,peso,valor);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

   

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	printf("\n");

	//Imprimindo a matriz
	for (i = 0; i<ITEN + 1; i++) {
		for (j = 0; j<WEI + 1; j++) {
			printf("%d ", mochila[(i*(WEI + 1)) + j]);

		}
		printf("\n"); // para pular linha quando terminar a coluna
	}


	printf("\n");


	printf("\n");
	printf("Valor maximo da mochila: %d\n", mochila[ITEN*(WEI + 1) + WEI]);

	free(mochila);
	system("pause");
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t mochilaWithCuda(int *host_mochila, const int *host_peso, const int *host_valor)
{
    int *dev_mochila = 0;
    int *dev_peso = 0;
    int *dev_valor = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_mochila,(ITEN + 1)*(WEI + 1)*sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_peso, ITEN * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_valor, ITEN * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_mochila, host_mochila, (ITEN + 1)*(WEI + 1)* sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_valor, host_valor, ITEN * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_peso, host_peso , ITEN * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


    // Launch a kernel on the GPU with one thread for each element.
    mochilaKernel<<<1, WEI>>>(dev_mochila, dev_peso, dev_valor, WEI, ITEN);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(host_mochila, dev_mochila, (ITEN+1)*(WEI+1)* sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_mochila);
    hipFree(dev_valor);
    hipFree(dev_peso);
    
    return cudaStatus;
}

void inserirPeso(int vet[]){
	vet[1] = 1;
	vet[2] = 2;
	vet[3] = 5;
	vet[4] = 6;
	vet[5] = 7;
}

void inserirValor(int vet[]){
	vet[1] = 1;
	vet[2] = 6;
	vet[3] = 18;
	vet[4] = 22;
	vet[5] = 28;
}






void info(int tam, int pes[], int val[], int n){
	int i;
	printf("=========================================================\n");
	printf("                  *Dados da mochila*                     \n");
	printf("\n");
	printf("Capacidade total da mochila: %d\n", tam);
	printf("Numero de itens: %d itens", n);
	printf("\n");

	printf("Valor de cada item: ");
	for (i = 1; i<n + 1; i++){
		printf("%d ", val[i]);
	}

	printf("\n");

	printf("Peso de cada item: ");
	for (i = 1; i<n + 1; i++){
		printf("%d ", pes[i]);
	}



	printf("\n=========================================================\n");
}
