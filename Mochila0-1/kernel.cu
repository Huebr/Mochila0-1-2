
#ifndef __CUDACC__
#define __CUDACC__
#endif
#include "hip/hip_runtime.h"


#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include <string.h>

//#define WEI 11
//#define ITEN 5

//void inserirPeso(int vet[]);
//void inserirValor(int vet[]);


void info(int tam, int pes[], int val[], int n);
void encItens(int *mol, int n, int W, int pes[], int val[], int nit, int wit);

hipError_t mochilaWithCuda(int *mochila, const int *peso, const int *valor, int iten, int pes);

__global__ void mochilaKernel(int *mochila, const int *peso, const int *valor,const int wei,const int iten)
{
	int i;
	int w = threadIdx.x + 1;
	if (w<wei+1){
		for (i = 1; i<iten + 1; i++){
			if (peso[i]>w){
				mochila[i*(wei + 1) + w] = mochila[(i - 1)*(wei + 1) + w];

			}
			else{
				mochila[i*(wei+1)+w] = max(mochila[(i - 1)*(wei+1)+w], valor[i] + mochila[(i - 1)*(wei+1)+ w - peso[i]]);

			}
		}
	}
}

int main()
{
    
	char url[80];

	//declara��o do peso limite da mochila, o numero de itens e variavel auxiliar
	int W, n, aux, aux2, q, i, j, *peso, *valor, *mochila;

	//C:/Users/Pedro/Downloads/test.in

	//---------------------Leitura do arquivo-----------------------------------------------

	printf("Digite o local do arquivo: ");
	printf("\n");
	scanf("%s", &url);

	FILE *arq;

	arq = fopen(url, "r");
	if (arq == NULL)
		printf("Erro, nao foi possivel abrir o arquivo\n");
	else{
		fscanf(arq, "%d\n", &n);
		aux = n + 1;

		//Aloca��o
		peso = (int*)malloc(aux*sizeof(int));
		valor = (int*)malloc(aux*sizeof(int));
		if (peso == NULL || valor == NULL){
			perror("Erro de alocacao de memoria vetor peso ou vetor valor");
			exit(EXIT_FAILURE);
		}

		for (i = 1; i<aux; i++){
			fscanf(arq, "%d %d %d\n", &q, &valor[i], &peso[i]);
		}

		fscanf(arq, "%d\n", &W);
		aux2 = W + 1;

		mochila = (int*)malloc(aux*aux2*sizeof(int));
		
		if (mochila == NULL){
			perror("Nao foi possivel alocar a mochila");
			exit(EXIT_FAILURE);
		}
		for (i = 0; i < (aux)*(aux2); i++) mochila[i] = 0;

	}

	//Fecha o Arquivo
	fclose(arq);
	//------------------------Final do Arquvio------------------------------------------


	

	//inser��es dos itens com seus respectivos valores
	//inserirPeso(peso);
	//inserirValor(valor);

	info(W, peso, valor, n);

	//Pseudo Codigo Transcrito


    // Add vectors in parallel.
    hipError_t cudaStatus = mochilaWithCuda(mochila,peso,valor,n,W);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

   

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	printf("\n");

	encItens(mochila, n, W, peso, valor, n, W);

	printf("\n");
	printf("\n");

	//Imprimindo a matriz
	for (i = 0; i<n + 1; i++) {
		for (j = 0; j<W + 1; j++) {
			printf("%d ", mochila[(i*(W + 1)) + j]);

		}
		printf("\n"); // para pular linha quando terminar a coluna
	}


	printf("\n");


	printf("\n");
	printf("Valor maximo da mochila: %d\n", mochila[n*(W + 1) + W]);

	free(mochila);
	free(valor);
	free(peso);
	system("pause");
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t mochilaWithCuda(int *host_mochila, const int *host_peso, const int *host_valor, int iten, int pes)
{
    int *dev_mochila = 0;
    int *dev_peso = 0;
    int *dev_valor = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_mochila,(iten + 1)*(pes + 1)*sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_peso, (iten+1) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_valor, (iten+1) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_mochila, host_mochila, (iten + 1)*(pes + 1)* sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	
    cudaStatus = hipMemcpy(dev_valor, host_valor, (iten+1) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_peso, host_peso , (iten+1) * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


    // Launch a kernel on the GPU with one thread for each element.
    mochilaKernel<<<1, pes>>>(dev_mochila, dev_peso, dev_valor, pes, iten);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(host_mochila, dev_mochila, (iten+1)*(pes+1)* sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_mochila);
    hipFree(dev_valor);
    hipFree(dev_peso);
    
    return cudaStatus;
}

void inserirPeso(int vet[]){
	vet[1] = 1;
	vet[2] = 2;
	vet[3] = 5;
	vet[4] = 6;
	vet[5] = 7;
}

void inserirValor(int vet[]){
	vet[1] = 1;
	vet[2] = 6;
	vet[3] = 18;
	vet[4] = 22;
	vet[5] = 28;
}






void info(int tam, int pes[], int val[], int n){
	int i;
	printf("=========================================================\n");
	printf("                  *Dados da mochila*                     \n");
	printf("\n");
	printf("Capacidade total da mochila: %d\n", tam);
	printf("Numero de itens: %d itens", n);
	printf("\n");

	printf("Valor de cada item: ");
	for (i = 1; i<n + 1; i++){
		printf("%d ", val[i]);
	}

	printf("\n");

	printf("Peso de cada item: ");
	for (i = 1; i<n + 1; i++){
		printf("%d ", pes[i]);
	}



	printf("\n=========================================================\n");
}

void encItens(int *mol, int n, int W, int pes[], int val[], int nit, int wit){

	int i, pcal;


	//mat[n] abc

	if (mol[(n*(wit+1))+W] == mol[((n - 1)*(wit+1))+W]){
		if (mol[(n*(wit+1)) + W] != 0){
			encItens(mol, n - 1, W, pes, val, nit, wit);
		}
	}
	else{

		for (i = 1; i<nit + 1; i++){
			pcal = W - pes[i];
			if (val[i] + mol[((i - 1)*(wit+1)) + pcal] == mol[(n*(wit+1)) + W]){
				printf("%d ", i);
				encItens(mol, i - 1, pcal, pes, val, nit, wit);
				break;
			}
			if (mol[(n*(wit+1)) + W] == val[i]){
				printf("%d ", i);

				break;
			}


		}

	}



}
